#include <gtsam_points/cuda/gl_buffer_map.hpp>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <gtsam_points/cuda/check_error.cuh>

namespace gtsam_points {

using gtsam_points::check_error;

GLBufferMap::GLBufferMap(size_t gl_id, BufferMode mode) {
  unsigned int flags = 0;

  switch (mode) {
    case GLBufferMap::NONE:
      flags = hipGraphicsRegisterFlagsNone;
      break;
    case GLBufferMap::READ_ONLY:
      flags = hipGraphicsRegisterFlagsReadOnly;
      break;
    case GLBufferMap::WRITE_ONLY:
      flags = hipGraphicsRegisterFlagsWriteDiscard;
      break;
  }

  check_error << hipGraphicsGLRegisterBuffer(&gl_resource, gl_id, flags);
  check_error << hipGraphicsMapResources(1, &gl_resource);
  check_error << hipGraphicsResourceGetMappedPointer(&d_buffer, &buffer_size, gl_resource);
}

GLBufferMap::~GLBufferMap() {
  check_error << hipGraphicsUnmapResources(1, &gl_resource);
  check_error << hipGraphicsUnregisterResource(gl_resource);
}

}  // namespace gtsam_points
