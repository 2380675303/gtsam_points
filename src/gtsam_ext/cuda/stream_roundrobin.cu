#include <glim/cuda/util/stream_roundrobin.hpp>

#include <hip/hip_runtime_api.h>

namespace gtsam_ext {

StreamRoundRobin::StreamRoundRobin(int num_streams) {
  streams.resize(num_streams);
  for (int i = 0; i < num_streams; i++) {
    hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
  }
  cursor = 0;
}

StreamRoundRobin::~StreamRoundRobin() {
  for (int i = 0; i < streams.size(); i++) {
    hipStreamSynchronize(streams[i]);
    hipStreamDestroy(streams[i]);
  }
}

void StreamRoundRobin::sync_all() {
  for (int i = 0; i < streams.size(); i++) {
    hipStreamSynchronize(streams[i]);
  }
}

hipStream_t StreamRoundRobin::get_stream() {
  int stream = cursor++;
  return streams[stream % streams.size()];
}

}  // namespace gtsam_ext