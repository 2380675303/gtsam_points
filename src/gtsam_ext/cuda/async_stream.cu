#include <gtsam_ext/cuda/async_stream.hpp>

#include <iostream>

namespace gtsam_ext {

AsyncStream::AsyncStream() {
  hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
}

AsyncStream::~AsyncStream() {
  hipStreamSynchronize(stream);

  for (auto itr = resources.rbegin(); itr != resources.rend(); itr++) {
    itr->reset();
  }

  hipStreamDestroy(stream);
}

void AsyncStream::sync() {
  hipStreamSynchronize(stream);
}

void AsyncStream::add_resource(const std::shared_ptr<void>& resource) {
  resources.push_back(resource);
}

RegisteredMemory::RegisteredMemory(void* ptr, size_t size, unsigned int flags) {
  hipHostRegister(ptr, size, flags);
}

RegisteredMemory::~RegisteredMemory() {
  hipHostUnregister(ptr);
}

}  // namespace gtsam_ext