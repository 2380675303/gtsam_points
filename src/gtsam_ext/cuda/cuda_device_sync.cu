// SPDX-License-Identifier: MIT
// Copyright (c) 2021  Kenji Koide (k.koide@aist.go.jp)

#include <gtsam_ext/cuda/cuda_device_sync.hpp>

namespace gtsam_ext {

void cuda_device_synchronize() {
  hipDeviceSynchronize();
}

}