#include <gtsam_ext/cuda/hip_stream.hpp>

#include <iostream>
#include <gtsam_ext/cuda/check_error.cuh>

namespace gtsam_ext {

CUDAStream::CUDAStream() {
  check_error << hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
}

CUDAStream::~CUDAStream() {
  check_error << hipStreamSynchronize(stream);

  for (auto itr = resources.rbegin(); itr != resources.rend(); itr++) {
    itr->reset();
  }

  check_error << hipStreamDestroy(stream);
}

void CUDAStream::sync() {
  check_error << hipStreamSynchronize(stream);
}

void CUDAStream::add_resource(const std::shared_ptr<void>& resource) {
  resources.push_back(resource);
}

RegisteredMemory::RegisteredMemory(void* ptr, size_t size, unsigned int flags) {
  check_error << hipHostRegister(ptr, size, flags);
}

RegisteredMemory::~RegisteredMemory() {
  check_error << hipHostUnregister(ptr);
}

}  // namespace gtsam_ext