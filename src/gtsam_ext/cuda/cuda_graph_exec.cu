// SPDX-License-Identifier: MIT
// Copyright (c) 2021  Kenji Koide (k.koide@aist.go.jp)

#include <gtsam_ext/cuda/cuda_graph_exec.hpp>

#include <gtsam_ext/cuda/check_error.cuh>

namespace gtsam_ext {

CUDAGraphExec::CUDAGraphExec(ihipGraph* graph) {
  check_error << hipGraphInstantiate(&instance, graph, nullptr, nullptr, 0);
}

CUDAGraphExec::~CUDAGraphExec() {
  check_error << hipGraphExecDestroy(instance);
}

void CUDAGraphExec::launch(ihipStream_t* stream) {
  check_error << hipGraphLaunch(instance, stream);
}

}  // namespace gtsam_ext