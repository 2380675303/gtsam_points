// SPDX-License-Identifier: MIT
// Copyright (c) 2021  Kenji Koide (k.koide@aist.go.jp)

#include <gtsam_ext/types/frame_gpu.hpp>

#include <thrust/transform.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <gtsam_ext/types/cpu_gpu_copy.hpp>

namespace gtsam_ext {

template <typename T, int D>
FrameGPU::FrameGPU(const Eigen::Matrix<T, D, 1>* points, int num_points, bool allocate_cpu)
: times_gpu_storage(new FloatsGPU()),
  points_gpu_storage(new PointsGPU()),
  normals_gpu_storage(new PointsGPU()),
  covs_gpu_storage(new MatricesGPU()) {
  //
  if (allocate_cpu) {
    add_points(points, num_points);
  } else {
    add_points_gpu(points, num_points);
  }
}

template <typename T, int D>
FrameGPU::FrameGPU(const std::vector<Eigen::Matrix<T, D, 1>, Eigen::aligned_allocator<Eigen::Matrix<T, D, 1>>>& points, bool allocate_cpu)
: FrameGPU(points.data(), points.size(), allocate_cpu) {}

FrameGPU::FrameGPU(const Frame& frame, bool allocate_cpu)
: times_gpu_storage(new FloatsGPU()),
  points_gpu_storage(new PointsGPU()),
  normals_gpu_storage(new PointsGPU()),
  covs_gpu_storage(new MatricesGPU()) {
  //
  num_points = frame.size();

  if (allocate_cpu) {
    if (frame.points) {
      add_points(frame.points, frame.size());
    }

    if (frame.times) {
      add_times(frame.times, frame.size());
    }

    if (frame.normals) {
      add_normals(frame.normals, frame.size());
    }

    if (frame.covs) {
      add_covs(frame.covs, frame.size());
    }
  } else {
    if (frame.points) {
      add_points_gpu(frame.points, frame.size());
    }

    if (frame.times) {
      add_times_gpu(frame.times, frame.size());
    }

    if (frame.normals) {
      add_normals_gpu(frame.normals, frame.size());
    }

    if (frame.covs) {
      add_covs_gpu(frame.covs, frame.size());
    }
  }

  /*
  copy_to_gpu(*times_gpu_storage, &times_gpu, frame.times, frame.times_gpu, num_points);
  copy_to_gpu(*points_gpu_storage, &points_gpu, frame.points, frame.points_gpu, num_points);
  copy_to_gpu(*normals_gpu_storage, &normals_gpu, frame.normals, frame.normals_gpu, num_points);
  copy_to_gpu(*covs_gpu_storage, &covs_gpu, frame.covs, frame.covs_gpu, num_points);

  if (allocate_cpu) {
    copy_to_cpu(times_storage, &times, frame.times, frame.times_gpu, num_points, 0.0);
    copy_to_cpu(points_storage, &points, frame.points, frame.points_gpu, num_points, Eigen::Vector4d(0.0, 0.0, 0.0, 1.0));
    copy_to_cpu(normals_storage, &normals, frame.normals, frame.normals_gpu, num_points, Eigen::Vector4d::Zero().eval());
    copy_to_cpu(covs_storage, &covs, frame.covs, frame.covs_gpu, num_points, Eigen::Matrix4d::Zero().eval());
  }
  */
}

FrameGPU::FrameGPU() : times_gpu_storage(new FloatsGPU()), points_gpu_storage(new PointsGPU()), normals_gpu_storage(new PointsGPU()), covs_gpu_storage(new MatricesGPU()) {}

FrameGPU::~FrameGPU() {}

template <typename T>
void FrameGPU::add_times(const T* times, int num_points) {
  add_times_gpu(times, num_points);

  times_storage.resize(num_points);
  thrust::copy(times, times + num_points, times_storage.begin());
  this->times = times_storage.data();
}

template <typename T>
void FrameGPU::add_times_gpu(const T* times, int num_points) {
  assert(num_points == size());
  thrust::host_vector<float> times_h(num_points);
  std::copy(times, times + num_points, times_h.begin());

  times_gpu_storage->resize(num_points);
  hipMemcpy(thrust::raw_pointer_cast(times_gpu_storage->data()), times_h.data(), sizeof(float) * num_points, hipMemcpyHostToDevice);
  this->times_gpu = thrust::raw_pointer_cast(times_gpu_storage->data());
}

template <typename T>
void FrameGPU::add_times(const std::vector<T>& times) {
  add_times(times.data(), times.size());
}

template <typename T>
void FrameGPU::add_times_gpu(const std::vector<T>& times) {
  add_times_gpu(times.data(), times.size());
}

template <typename T, int D>
void FrameGPU::add_points(const Eigen::Matrix<T, D, 1>* points, int num_points) {
  add_points_gpu(points, num_points);

  points_storage.resize(num_points, Eigen::Vector4d::UnitW());
  this->points = points_storage.data();
  for (int i = 0; i < num_points; i++) {
    points_storage[i].template head<D>() = points[i].template cast<double>();
  }
}

template <typename T, int D>
void FrameGPU::add_points_gpu(const Eigen::Matrix<T, D, 1>* points, int num_points) {
  this->num_points = num_points;
  points_gpu_storage->resize(num_points);
  points_gpu = thrust::raw_pointer_cast(points_gpu_storage->data());

  std::vector<Eigen::Vector3f, Eigen::aligned_allocator<Eigen::Vector3f>> points_f(num_points);
  for (int i = 0; i < num_points; i++) {
    points_f[i] = points[i].template head<3>().template cast<float>();
  }
  hipMemcpy(points_gpu, points_f.data(), sizeof(Eigen::Vector3f) * num_points, hipMemcpyHostToDevice);
}

template <typename T, int D>
void FrameGPU::add_points(const std::vector<Eigen::Matrix<T, D, 1>, Eigen::aligned_allocator<Eigen::Matrix<T, D, 1>>>& points) {
  add_points(points.data(), points.size());
}

template <typename T, int D>
void FrameGPU::add_points_gpu(const std::vector<Eigen::Matrix<T, D, 1>, Eigen::aligned_allocator<Eigen::Matrix<T, D, 1>>>& points) {
  add_points_gpu(points.data(), points.size());
}

template <typename T, int D>
void FrameGPU::add_normals(const Eigen::Matrix<T, D, 1>* normals, int num_points) {
  add_normals_gpu(normals, num_points);

  normals_storage.resize(num_points, Eigen::Vector4d(0, 0, 0, 0));
  for (int i = 0; i < num_points; i++) {
    normals_storage[i].template head<D>() = normals[i].template cast<double>();
  }
  this->normals = normals_storage.data();
}

template <typename T, int D>
void FrameGPU::add_normals_gpu(const Eigen::Matrix<T, D, 1>* normals, int num_points) {
  assert(normals.size() == size());

  thrust::host_vector<Eigen::Vector3f, Eigen::aligned_allocator<Eigen::Vector3f>> normals_h(num_points);
  for (int i = 0; i < num_points; i++) {
    normals_h[i] = normals[i].template head<3>().template cast<float>();
  }

  normals_gpu_storage->resize(num_points);
  hipMemcpy(thrust::raw_pointer_cast(normals_gpu_storage->data()), normals_h.data(), sizeof(Eigen::Vector3f) * num_points, hipMemcpyHostToDevice);
  this->normals_gpu = thrust::raw_pointer_cast(normals_gpu_storage->data());
}

template <typename T, int D>
void FrameGPU::add_normals(const std::vector<Eigen::Matrix<T, D, 1>, Eigen::aligned_allocator<Eigen::Matrix<T, D, 1>>>& normals) {
  add_normals(normals.data(), normals.size());
}

template <typename T, int D>
void FrameGPU::add_normals_gpu(const std::vector<Eigen::Matrix<T, D, 1>, Eigen::aligned_allocator<Eigen::Matrix<T, D, 1>>>& normals) {
  add_normals_gpu(normals.data(), normals.size());
}

template <typename T, int D>
void FrameGPU::add_covs(const Eigen::Matrix<T, D, D>* covs, int num_points) {
  add_covs_gpu(covs, num_points);

  covs_storage.resize(num_points, Eigen::Matrix4d::Zero());
  for (int i = 0; i < num_points; i++) {
    covs_storage[i].template block<D, D>(0, 0) = covs[i].template cast<double>();
  }
  this->covs = covs_storage.data();
}

template <typename T, int D>
void FrameGPU::add_covs_gpu(const Eigen::Matrix<T, D, D>* covs, int num_points) {
  assert(covs.size() == size());
  thrust::host_vector<Eigen::Matrix3f, Eigen::aligned_allocator<Eigen::Matrix3f>> covs_h(num_points);
  for (int i = 0; i < num_points; i++) {
    covs_h[i] = covs[i].template block<3, 3>(0, 0).template cast<float>();
  }

  covs_gpu_storage->resize(num_points);
  hipMemcpy(thrust::raw_pointer_cast(covs_gpu_storage->data()), covs_h.data(), sizeof(Eigen::Matrix3f) * num_points, hipMemcpyHostToDevice);
  this->covs_gpu = thrust::raw_pointer_cast(covs_gpu_storage->data());
}

template <typename T, int D>
void FrameGPU::add_covs(const std::vector<Eigen::Matrix<T, D, D>, Eigen::aligned_allocator<Eigen::Matrix<T, D, D>>>& covs) {
  add_covs(covs.data(), covs.size());
}

template <typename T, int D>
void FrameGPU::add_covs_gpu(const std::vector<Eigen::Matrix<T, D, D>, Eigen::aligned_allocator<Eigen::Matrix<T, D, D>>>& covs) {
  add_covs_gpu(covs.data(), covs.size());
}

template FrameGPU::FrameGPU(const Eigen::Matrix<float, 3, 1>*, int, bool);
template FrameGPU::FrameGPU(const Eigen::Matrix<float, 4, 1>*, int, bool);
template FrameGPU::FrameGPU(const Eigen::Matrix<double, 3, 1>*, int, bool);
template FrameGPU::FrameGPU(const Eigen::Matrix<double, 4, 1>*, int, bool);

template FrameGPU::FrameGPU(const std::vector<Eigen::Matrix<float, 3, 1>, Eigen::aligned_allocator<Eigen::Matrix<float, 3, 1>>>&, bool);
template FrameGPU::FrameGPU(const std::vector<Eigen::Matrix<float, 4, 1>, Eigen::aligned_allocator<Eigen::Matrix<float, 4, 1>>>&, bool);
template FrameGPU::FrameGPU(const std::vector<Eigen::Matrix<double, 3, 1>, Eigen::aligned_allocator<Eigen::Matrix<double, 3, 1>>>&, bool);
template FrameGPU::FrameGPU(const std::vector<Eigen::Matrix<double, 4, 1>, Eigen::aligned_allocator<Eigen::Matrix<double, 4, 1>>>&, bool);

template void FrameGPU::add_times(const std::vector<float>&);
template void FrameGPU::add_times(const std::vector<double>&);
template void FrameGPU::add_times_gpu(const std::vector<float>&);
template void FrameGPU::add_times_gpu(const std::vector<double>&);

template void FrameGPU::add_points(const std::vector<Eigen::Matrix<float, 3, 1>, Eigen::aligned_allocator<Eigen::Matrix<float, 3, 1>>>&);
template void FrameGPU::add_points(const std::vector<Eigen::Matrix<float, 4, 1>, Eigen::aligned_allocator<Eigen::Matrix<float, 4, 1>>>&);
template void FrameGPU::add_points(const std::vector<Eigen::Matrix<double, 3, 1>, Eigen::aligned_allocator<Eigen::Matrix<double, 3, 1>>>&);
template void FrameGPU::add_points(const std::vector<Eigen::Matrix<double, 4, 1>, Eigen::aligned_allocator<Eigen::Matrix<double, 4, 1>>>&);
template void FrameGPU::add_points_gpu(const std::vector<Eigen::Matrix<float, 3, 1>, Eigen::aligned_allocator<Eigen::Matrix<float, 3, 1>>>&);
template void FrameGPU::add_points_gpu(const std::vector<Eigen::Matrix<float, 4, 1>, Eigen::aligned_allocator<Eigen::Matrix<float, 4, 1>>>&);
template void FrameGPU::add_points_gpu(const std::vector<Eigen::Matrix<double, 3, 1>, Eigen::aligned_allocator<Eigen::Matrix<double, 3, 1>>>&);
template void FrameGPU::add_points_gpu(const std::vector<Eigen::Matrix<double, 4, 1>, Eigen::aligned_allocator<Eigen::Matrix<double, 4, 1>>>&);

template void FrameGPU::add_normals(const std::vector<Eigen::Matrix<float, 3, 1>, Eigen::aligned_allocator<Eigen::Matrix<float, 3, 1>>>&);
template void FrameGPU::add_normals(const std::vector<Eigen::Matrix<float, 4, 1>, Eigen::aligned_allocator<Eigen::Matrix<float, 4, 1>>>&);
template void FrameGPU::add_normals(const std::vector<Eigen::Matrix<double, 3, 1>, Eigen::aligned_allocator<Eigen::Matrix<double, 3, 1>>>&);
template void FrameGPU::add_normals(const std::vector<Eigen::Matrix<double, 4, 1>, Eigen::aligned_allocator<Eigen::Matrix<double, 4, 1>>>&);
template void FrameGPU::add_normals_gpu(const std::vector<Eigen::Matrix<float, 3, 1>, Eigen::aligned_allocator<Eigen::Matrix<float, 3, 1>>>&);
template void FrameGPU::add_normals_gpu(const std::vector<Eigen::Matrix<float, 4, 1>, Eigen::aligned_allocator<Eigen::Matrix<float, 4, 1>>>&);
template void FrameGPU::add_normals_gpu(const std::vector<Eigen::Matrix<double, 3, 1>, Eigen::aligned_allocator<Eigen::Matrix<double, 3, 1>>>&);
template void FrameGPU::add_normals_gpu(const std::vector<Eigen::Matrix<double, 4, 1>, Eigen::aligned_allocator<Eigen::Matrix<double, 4, 1>>>&);

template void FrameGPU::add_covs(const std::vector<Eigen::Matrix<float, 3, 3>, Eigen::aligned_allocator<Eigen::Matrix<float, 3, 3>>>&);
template void FrameGPU::add_covs(const std::vector<Eigen::Matrix<float, 4, 4>, Eigen::aligned_allocator<Eigen::Matrix<float, 4, 4>>>&);
template void FrameGPU::add_covs(const std::vector<Eigen::Matrix<double, 3, 3>, Eigen::aligned_allocator<Eigen::Matrix<double, 3, 3>>>&);
template void FrameGPU::add_covs(const std::vector<Eigen::Matrix<double, 4, 4>, Eigen::aligned_allocator<Eigen::Matrix<double, 4, 4>>>&);
template void FrameGPU::add_covs_gpu(const std::vector<Eigen::Matrix<float, 3, 3>, Eigen::aligned_allocator<Eigen::Matrix<float, 3, 3>>>&);
template void FrameGPU::add_covs_gpu(const std::vector<Eigen::Matrix<float, 4, 4>, Eigen::aligned_allocator<Eigen::Matrix<float, 4, 4>>>&);
template void FrameGPU::add_covs_gpu(const std::vector<Eigen::Matrix<double, 3, 3>, Eigen::aligned_allocator<Eigen::Matrix<double, 3, 3>>>&);
template void FrameGPU::add_covs_gpu(const std::vector<Eigen::Matrix<double, 4, 4>, Eigen::aligned_allocator<Eigen::Matrix<double, 4, 4>>>&);

// copy data from GPU to CPU
std::vector<Eigen::Vector3f, Eigen::aligned_allocator<Eigen::Vector3f>> FrameGPU::get_points_gpu() const {
  std::vector<Eigen::Vector3f, Eigen::aligned_allocator<Eigen::Vector3f>> points_h;
  if (!points_gpu) {
    return points_h;
  }

  points_h.resize(num_points);
  hipMemcpy(points_h.data(), points_gpu, sizeof(Eigen::Vector3f) * num_points, hipMemcpyDeviceToHost);
  return points_h;
}

std::vector<Eigen::Matrix3f, Eigen::aligned_allocator<Eigen::Matrix3f>> FrameGPU::get_covs_gpu() const {
  std::vector<Eigen::Matrix3f, Eigen::aligned_allocator<Eigen::Matrix3f>> covs_h;
  if (!covs_gpu) {
    return covs_h;
  }

  covs_h.resize(num_points);
  hipMemcpy(covs_h.data(), covs_gpu, sizeof(Eigen::Matrix3f) * num_points, hipMemcpyDeviceToHost);
  return covs_h;
}

}  // namespace gtsam_ext